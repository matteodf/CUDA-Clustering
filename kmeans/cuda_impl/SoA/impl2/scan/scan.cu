#include "hip/hip_runtime.h"
#include "scan.h"

void scan(int *d_out, int *d_in, int length) {
    if (length > ELEMENTS_PER_BLOCK) {
        scanLargeDeviceArray(d_out, d_in, length);
    } else {
        scanSmallDeviceArray(d_out, d_in, length);
    }

    return;
}

void scanLargeDeviceArray(int *d_out, int *d_in, int length) {
    int remainder = length % (ELEMENTS_PER_BLOCK);
    if (remainder == 0) {
        scanLargeEvenDeviceArray(d_out, d_in, length);
    } else {
        // perform a large scan on a compatible multiple of elements
        int lengthMultiple = length - remainder;
        scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple);

        // scan the remaining elements and add the (inclusive) last element of the large scan to this
        int *startOfOutputArray = &(d_out[lengthMultiple]);
        scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder);

        add<<<1, remainder>>>(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
    }
}

void scanSmallDeviceArray(int *d_out, int *d_in, int length) {
    int powerOfTwo = nextPowerOfTwo(length);
    prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
}

void scanLargeEvenDeviceArray(int *d_out, int *d_in, int length) {
    const int blocks = length / ELEMENTS_PER_BLOCK;
    const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

    int *d_sums, *d_incr;
    hipMalloc((void **)&d_sums, blocks * sizeof(int));
    hipMalloc((void **)&d_incr, blocks * sizeof(int));

    prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);

    const int sumsArrThreadsNeeded = (blocks + 1) / 2;
    if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
        // perform a large scan on the sums arr
        scanLargeDeviceArray(d_incr, d_sums, blocks);
    } else {
        // only need one block to scan sums arr so can use small scan
        scanSmallDeviceArray(d_incr, d_sums, blocks);
    }

    add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

    hipFree(d_sums);
    hipFree(d_incr);
}

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo) {
    extern __shared__ int temp[];  // allocated on invocation
    int threadID = threadIdx.x;

    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    if (threadID < n) {
        temp[ai + bankOffsetA] = input[ai];
        temp[bi + bankOffsetB] = input[bi];
    } else {
        temp[ai + bankOffsetA] = 0;
        temp[bi + bankOffsetB] = 0;
    }

    int offset = 1;
    for (int d = powerOfTwo >> 1; d > 0; d >>= 1)  // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d) {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (threadID == 0) {
        temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0;  // clear the last element
    }

    for (int d = 1; d < powerOfTwo; d *= 2)  // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d) {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (threadID < n) {
        output[ai] = temp[ai + bankOffsetA];
        output[bi] = temp[bi + bankOffsetB];
    }
}

__global__ void prescan_large(int *output, int *input, int n, int *sums) {
    extern __shared__ int temp[];

    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * n;

    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    temp[ai + bankOffsetA] = input[blockOffset + ai];
    temp[bi + bankOffsetB] = input[blockOffset + bi];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1)  // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d) {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    __syncthreads();

    if (threadID == 0) {
        sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
        temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
    }

    for (int d = 1; d < n; d *= 2)  // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d) {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    output[blockOffset + ai] = temp[ai + bankOffsetA];
    output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void add(int *output, int length, int *n) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}

int nextPowerOfTwo(int x) {
    int power = 1;
    while (power < x) {
        power *= 2;
    }
    return power;
}