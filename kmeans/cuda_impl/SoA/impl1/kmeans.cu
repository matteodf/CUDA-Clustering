#include "hip/hip_runtime.h"

#include "kmeans_utils.h"
#include <string>

#define NUMBER_OF_POINTS (1024*1024)
#define NUMBER_OF_CLUSTERS 32
#define MAXIMUM_ITERATIONS 100
#define RADIUS 100.0

#define PRINT_RESULTS false
#define READ_FROM_FILE true

#define THREADSxBLOCK 512

__constant__ float const_cx[NUMBER_OF_CLUSTERS];
__constant__ float const_cy[NUMBER_OF_CLUSTERS];

__global__ void addUpClusters(POINT pts, int numPts, POINT centroids) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numPts) {
        int g = pts.group[tid];
        atomicAdd(&centroids.group[g], 1);
        atomicAdd(&centroids.x[g], pts.x[tid]);
        atomicAdd(&centroids.y[g], pts.y[tid]);
    }
}

__global__ void computeMean(POINT centroids, int numCentroids) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numCentroids) {
        centroids.x[tid] /= centroids.group[tid];
        centroids.y[tid] /= centroids.group[tid];
    }
}

void updateCentroids(POINT d_pts, int numPts, POINT d_centroids, int numCentroids) {
    int threads = THREADSxBLOCK;
    int blocks = (numPts + threads - 1) / threads;
    CHECK(hipMemset(d_centroids.group, 0, sizeof(int) * numCentroids));
    CHECK(hipMemset(d_centroids.x, 0, sizeof(float) * numCentroids));
    CHECK(hipMemset(d_centroids.y, 0, sizeof(float) * numCentroids));
    addUpClusters<<<blocks, threads>>>(d_pts, numPts, d_centroids);
    CHECK(hipDeviceSynchronize());
    computeMean<<<(numCentroids + threads - 1) / threads, threads>>>(d_centroids, numCentroids);
    CHECK(hipDeviceSynchronize());
}

__global__ void updatePoints(int *changes, POINT pts, int numPts, int numCentroids) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float d, min_d, px, py;
    int j, clusterIndex;

    min_d = HUGE_VAL;
    if (tid < numPts) {
        clusterIndex = pts.group[tid];
        px = pts.x[tid];
        py = pts.y[tid];
        for (j = 0; j < numCentroids; j++) {
            d = dist2(const_cx[j], const_cy[j], px, py);
            if (d < min_d) {
                min_d = d;
                clusterIndex = j;
            }
        }
        if (clusterIndex != pts.group[tid]) {
            pts.group[tid] = clusterIndex;
            atomicAdd(changes, 1);
        }
    }
}

int kmeans(POINT pts, int numPts, POINT centroids, int numCentroids, int maxTimes) {
    int acceptable = numPts / 1000;

    POINT d_pts;
    POINT d_centroids;

    CHECK(hipMalloc((void **)&(d_pts.x), sizeof(float) * numPts));
    CHECK(hipMalloc((void **)&(d_pts.y), sizeof(float) * numPts));
    CHECK(hipMalloc((void **)&(d_pts.group), sizeof(int) * numPts));

    CHECK(hipMalloc((void **)&(d_centroids.x), sizeof(float) * numCentroids));
    CHECK(hipMalloc((void **)&(d_centroids.y), sizeof(float) * numCentroids));
    CHECK(hipMalloc((void **)&(d_centroids.group), sizeof(int) * numCentroids));

    int h_changes;
    int *d_changes;
    CHECK(hipMalloc((void **)&d_changes, sizeof(int)));

    int threads = THREADSxBLOCK;
    int blocks = (numPts + threads - 1) / threads;

    cudaMemcpyPS(d_pts, pts, numPts, hipMemcpyHostToDevice);
    cudaMemcpyPS(d_centroids, centroids, numCentroids, hipMemcpyHostToDevice);

    do {
        updateCentroids(d_pts, numPts, d_centroids, numCentroids);
        h_changes = 0;
        CHECK(hipMemcpy(d_changes, &h_changes, sizeof(int), hipMemcpyHostToDevice));

        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_cx), d_centroids.x, sizeof(int) * numCentroids, 0, hipMemcpyDeviceToDevice));
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_cy), d_centroids.y, sizeof(int) * numCentroids, 0, hipMemcpyDeviceToDevice));

        updatePoints<<<blocks, threads>>>(d_changes, d_pts, numPts, numCentroids);
        CHECK(hipMemcpy(&h_changes, d_changes, sizeof(int), hipMemcpyDeviceToHost));
        maxTimes--;
    } while ((h_changes > acceptable) && (maxTimes > 0));

    cudaMemcpyPS(pts, d_pts, numPts, hipMemcpyDeviceToHost);
    cudaMemcpyPS(centroids, d_centroids, numCentroids, hipMemcpyDeviceToHost);

    for (int i = 0; i < numCentroids; i++)
        centroids.group[i] = i;

    cudaFreePS(d_pts);
    cudaFreePS(d_centroids);
    return (MAXIMUM_ITERATIONS - maxTimes);
}

void test(char* inputFile, bool initRand,int testIter){
    int numPts, numCentroids, maxTimes, numIter;
    double start, stop, timeInit, timeKmeans;
    
    numPts = findNumRows(inputFile);
    numCentroids = NUMBER_OF_CLUSTERS;
    maxTimes = MAXIMUM_ITERATIONS;

    POINT pts, centroids;
    mallocPS(&pts, numPts);
    mallocPS(&centroids, numCentroids);

    if (numCentroids == 1 || numPts <= 0 || numCentroids > numPts) {
        printf("Error, wrong parameters\n");
        exit(1);
    }
    if (maxTimes < 1) maxTimes = 1;
    
    readPointsFromFile(&pts, inputFile);
    
    for (int i=0; i < testIter; i++){
        start = seconds();
        if (initRand) initClustersRandom(pts, numPts, numCentroids);
        if (!initRand) initClusters(pts, numPts, centroids, numCentroids);
        stop = seconds();
        timeInit = stop - start;

        start = seconds();
        numIter = kmeans(pts, numPts, centroids, numCentroids, maxTimes);
        stop = seconds();
        timeKmeans = stop - start;

        printf("%f %f %d\n", timeInit, timeKmeans, numIter);
    }

    char outputFile[] = "../../../../output/output_impl1.txt";
    writePointsToFile(outputFile, pts, numPts, centroids, numCentroids);

    freePS(&pts);
    freePS(&centroids);
}

int main(int argc, char **argv) {
    if (argc != 4) {
        printf("Please follow this format: ./app [intputFile] [initRand = t / f] [numTestIterations]\n");
        return 0;
    }
    //char filename[] = "../../../../input/input_rand2.txt";
    char *filename = argv[1];
    bool initRand = (((std::string)argv[2]) == (std::string)"t");
    int testIter = std::stoi(argv[3]);

    test(filename, initRand, testIter);
    return 0;
}